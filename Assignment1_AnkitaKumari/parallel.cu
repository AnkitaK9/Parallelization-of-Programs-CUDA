#include "hip/hip_runtime.h"
%%cu 
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#define N_Threads 1024
#define N 1024


__global__ void vectoradd(int *arr1, int *arr2,int R)
{
  int id = blockIdx.x*blockDim.x + threadIdx.x;
    int s1=0;
    int s2=0;
 
   
	  for (int r=1;r<=R;r++)
	  {
	       if((id-r)>0 && (id-r)<N)
	      {
	      	  s1=s1+arr1[id-r];

	      }
	      	if((id+r)>0 && (id+r)<N)
	      {
	      	  s2=s2+arr1[id+r];
	      }
     arr2[id]=s1+s2;
    
   }
}

int main(void)
{
  struct timeval begin, end;
 
  int A[N], B[N], *d_A, *d_B;
   int R=32;
   for (int i=0;i<N;i++)
   {
       A[i]=i;
   }
 
 
   hipMalloc(&d_A, N * sizeof(int));
   hipMalloc(&d_B, N * sizeof(int));
   
   hipMemcpy(d_A,A,N*(sizeof(int)),hipMemcpyHostToDevice);

    int NoBlocks=N/N_Threads;
     gettimeofday(&begin, 0);
 
    vectoradd<<<NoBlocks, N_Threads>>>(d_A,d_B,R);
    gettimeofday(&end, 0);
 
    long seconds = end.tv_sec - begin.tv_sec;
    long microseconds = end.tv_usec - begin.tv_usec;
    double elapsed = seconds + microseconds*1e-6;
 
    
    hipDeviceSynchronize();
    hipMemcpy(B,d_B,N*(sizeof(int)),hipMemcpyDeviceToHost);
    
    
    for (int i = 0; i < N; i++)  
    {
     
      printf("%d\n", B[i]); 
    }
 printf("Time measured: %.6f seconds.\n", elapsed);

    hipFree(d_A);
    hipFree(d_B);
  
  }

#include "hip/hip_runtime.h"
%%cu
#include <stdio.h>  
#include <hip/hip_runtime.h>  
#include <sys/time.h>
#define tile_size 32
#define m 512


__global__ void addition(int *N_matrix,int *M_matrix)
{
    
    int i = (blockIdx.x*tile_size)+threadIdx.x;
    int j= (blockIdx.y*tile_size)+threadIdx.y;
    int matrix1;
    int matrix2;
             if ((i-1)<0)
            {
                matrix1=M_matrix[(i+1)*m + j];
            } 
            else if ((i+1)>=m)
            {
                matrix1=M_matrix[(i-1)*m + j];
            }
            else
            {
                matrix1=M_matrix[(i+1)*m + j]+M_matrix[(i-1)*m + j];
            }

            if ((j-1)<0)
            {
                matrix2=M_matrix[i*m + (j+1)];
            } 
            else if ((j+1)>=m)
            {
                matrix2=M_matrix[i*m + (j-1)];
            }
            else
            {
                matrix2=M_matrix[i*m + (j+1)]+M_matrix[i*m + (j-1)];
            }
            N_matrix[i*m +j]=matrix1+matrix2;
      
}

int main() {
    
   
     dim3 block(tile_size,tile_size);
     dim3 grid_size(m/tile_size, m/tile_size);
     struct timeval start, stop;

  int M[m][m],N[m][m];
  int *gpu_M,*gpu_N;
  int k=0;
  int l=0;
  for (k=0;k<m;k++)
  {
      for (l=0;l<m;l++)
      {
        M[k][l]=k+l;
      }
  }
  int size=m*m*sizeof(int);
  hipMalloc(&gpu_M, size);
  hipMalloc(&gpu_N, size);
  
   hipMemcpy(gpu_M,M,size, hipMemcpyHostToDevice);

  gettimeofday(&start, 0);
  addition<<<grid_size, block>>>(gpu_N,gpu_M);

  gettimeofday(&stop, 0);
    long seconds = stop.tv_sec - start.tv_sec;
    long microseconds = stop.tv_usec - start.tv_usec;
    double elapsedTime = seconds + microseconds*1e-6;

  hipMemcpy(N, gpu_N, m*m*sizeof(int), hipMemcpyDeviceToHost);  
   int x=0;
   int y=0;
  for (x=0;x<m;x++){
      for (y=0;y<m;y++){
        printf("%d ", N[x][y]);
      }
      printf("\n\n");
  }

  printf("Time taken: %.6f seconds.\n", elapsedTime);

  return 0;
}

#include "hip/hip_runtime.h"
%%cu
#include <stdio.h>  
#include <hip/hip_runtime.h>  
#include <sys/time.h>
#define m 4

__global__ void addition(int *N_matrix,int *M_matrix){
    int id = blockIdx.x*blockDim.x +threadIdx.x;
    int matrix1 ;
    int matrix2;
    int i = id/m;
    int j= id%m;
            if ((i-1)<0)
            {
                matrix1=M_matrix[(i+1)*m + j];
            } 
            else if ((i+1)>=m)
            {
                matrix1=M_matrix[(i-1)*m + j];
            }
            else
            {
                matrix1=M_matrix[(i+1)*m + j]+M_matrix[(i-1)*m + j];
            }

            if ((j-1)<0)
            {
                matrix2=M_matrix[i*m + (j+1)];
            } 
            else if ((j+1)>=m)
            {
                matrix2=M_matrix[i*m + (j-1)];
            }
            else
            {
                matrix2=M_matrix[i*m + (j+1)]+M_matrix[i*m + (j-1)];
            }
            N_matrix[i*m +j]=matrix1+matrix2;
      
}

int main() 
{
  struct timeval begin, end;
  int M[m][m], N[m][m],*d_M,*d_N;
  int k=0;
  int l=0;
  for (k=0;k<m;k++)
  { 
      for (l=0;l<m;l++)
      {
          M[k][l]=k+l;
      }
        
  }
  int matrixsize=m*m;
  int size=m*m*sizeof(int);
  hipMalloc(&d_M, size);
  hipMalloc(&d_N, size);
  hipMemcpy(d_M,M,size, hipMemcpyHostToDevice);
  gettimeofday(&begin, 0);
  addition<<<32, matrixsize>>>(d_N,d_M);
    gettimeofday(&end, 0);
    long seconds = end.tv_sec - begin.tv_sec;
    long microseconds = end.tv_usec - begin.tv_usec;
     double elapsed = seconds + microseconds*1e-6;
  hipMemcpy(N,d_N,size,hipMemcpyDeviceToHost);  
  int x=0;
  int y=0;
  for (x=0;x<m;x++)
  {
      for (y=0;y<m;y++)
      {
        printf("%d ", N[x][y]);
      }
      printf("\n\n");
  }
  printf("Time measured: %.6f seconds.\n", elapsed);
 
  return 0;
}

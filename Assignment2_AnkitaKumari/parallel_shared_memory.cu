#include "hip/hip_runtime.h"
%%cu
#include <stdio.h>  
#include <hip/hip_runtime.h>  
#include <sys/time.h>
#define tile_size 32
#define m 128

__global__ void addition(int *N_matrix,int *M_matrix){
    __shared__ int var[tile_size][tile_size];
    int matrix1, matrix2;
    int i =blockIdx.x*tile_size+threadIdx.x;
    int j= blockIdx.y*tile_size+threadIdx.y;
    int a=threadIdx.x;
    int b=threadIdx.y;

    for (int n= 0; n < m/tile_size; n++) 
    {
        var[a][b] = M_matrix[i*m + j];
   
    }
            while (a==0)
            {
                if ((i-1)<0)
                {
                  matrix1=var[(a+1)][b];
                } 
                else if ((i+1)>=m)
                {
                  matrix1=M_matrix[(i-1)*m + j];
                }
                else
                {
                  matrix1=var[(a+1)][b]+M_matrix[(i-1)*m + j];
                }
             } 

           while (b==0){
                if ((j-1)<0)
                {
                  matrix2=var[a][(b+1)];
                } 
                else if ((j+1)>=m)
                {
                  matrix2=M_matrix[i*m + (j-1)];
                }
                else
                {
                  matrix2=var[a][(b+1)]+M_matrix[i*m + (j-1)];;
                }  
      
}

int main() 
{
    
dim3 grid_size(m/tile_size, m/tile_size);
dim3 tile(tile_size,tile_size);
struct timeval start, stop;
int M[m][m], N[m][m];
int *gpu_M,*gpu_N;
int k=0;
int l=0;
  for (k=0;k<m;k++)
  {
      for (l=0;l<m;l++)
        {
        M[k][l]=k+l;
        }
  }
  int size=m*m*sizeof(int);
  hipMalloc(&gpu_M, size);
  hipMalloc(&gpu_N, size);
  hipMemcpy(gpu_M,M,size,hipMemcpyHostToDevice);

  gettimeofday(&start, 0);
  addition<<<grid_size, tile>>>(gpu_N,gpu_M);

  gettimeofday(&stop, 0);
    long seconds = stop.tv_sec - start.tv_sec;
    long microseconds = stop.tv_usec - start.tv_usec;
    double elapsedTime = seconds + microseconds*1e-6;

  hipMemcpy(N, gpu_N, size, hipMemcpyDeviceToHost);  
  int x=0;
  int y=0;
  for (x=0;x<m;x++)
  {
      for (y=0;y<m;y++)
      {
        printf("%d ", N[x][y]);
      }
      printf("\n\n");
  }

  printf("Time taken: %.6f seconds.\n", elapsedTime);

 
  return 0;
}
